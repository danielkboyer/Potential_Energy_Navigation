#include "hip/hip_runtime.h"
#include "Agent.h"
#include "Map.h"
#include "Properties.h"
#include "math.h"
#include "Util.h"
#include "GPU_Util.h"



void GPU_Util::StepAll(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map){
    for(int x = 0;x<inCount;x++){
		int aIndex = x*properties.numberOfDirectionSpawn;
		for(int y = 0;y<properties.numberOfDirectionSpawn;y++){
			float newDirection = in[x].direction - properties.directionSpawnRadius/2 + properties.directionSpawnRadius/(properties.numberOfDirectionSpawn-1) * y;
			//a[aIndex+y] = Agent();
			out[aIndex+y] = AgentStep(in[x],newDirection,properties,map);		
			//printf("Agent position %f,%f\n",a[aIndex+y].positionX,a[aIndex+y].positionY);
		}
	}
}
void GPU_Util::RandPrune(Agent* agents, long numberAgents, long agentsToPrune){
    srand (100);
    long x = 0;
    for(int i=0;i<agentsToPrune;i++){
        x = rand()%numberAgents;
        agents[x].pruned = true;
        //printf("ID pruned: %ld\n", x);
    }
}
//this will be done in serial
void GPU_Util::CalcAvg(Agent* agents, Properties properties, long sampleRate, Stat out, long numberAgents, long agentsToPrune){
    // get list of random number to interate through the agents 
    int randArrayIDs[sampleRate]; // array of ID's of agents
    //printf("\n randArrayIDs:");
    for(int i=0;i<sampleRate;i++){
        randArrayIDs[i]=rand()%numberAgents;  //Generate number between 0 to number of agents
        //printf("  %i  ",randArrayIDs[i]);
    }
    // make arrays of agent energies and distances for averaging
    float randDistances[sampleRate];
    float randEnergies[sampleRate];
    for (int i=0;i<sampleRate;i++){
        randDistances[i] = agents[randArrayIDs[i]].DistanceFrom(properties.agentStartX,properties.agentStartY);
        randEnergies[i] = agents[randArrayIDs[i]].Energy(properties.gravity,properties.friction);
        //printf("Rand Distances %d: %f\n",i,randDistances[i]);
        //printf("randEnergies %d: %f\n",i,randEnergies[i]);
        //printf("i, for agents %d: x  %f  y  %f\n",i,agents[randArrayIDs[i]].positionX, agents[randArrayIDs[i]].positionY);

    }
    // get average distance and average energy for each random ID
    for (int i=0;i<sampleRate;i++){
        //printf("Rand Distances %d: %f\n",i,out.d_avg);
        out.d_avg += randDistances[i]/(float(sampleRate));
        out.E_avg += randEnergies[i]/(float(sampleRate));
    }
    // get the normalized average
    float normalized[sampleRate];
    float avg_normalized = 0;
    for (int i=0;i<sampleRate;i++){
        //TODO: out.d_avg/out.d_avg equals 1 right?
        normalized[i] = sqrt(randDistances[i]*randDistances[i]/(out.d_avg*out.d_avg) + randEnergies[i]*randEnergies[i]/(out.E_avg*out.E_avg));
        avg_normalized +=normalized[i];
    }
    avg_normalized = avg_normalized/float(sampleRate);
    // calcualte the standard deviation
    float stdDeviation = 0;
    for (int i=0;i<sampleRate;i++){
        stdDeviation += (normalized[i] - avg_normalized)*(normalized[i] - avg_normalized);
    }
    stdDeviation = sqrt(stdDeviation/sampleRate);
    out.offset = avg_normalized + (-0.5 + float(agentsToPrune)/float(numberAgents))*5.0*stdDeviation + stdDeviation/10.0;
}


void GPU_Util::Prune(Agent* agents, int count, Properties properties, Stat stat){
    for(int x = 0;x<count;x++){
        CheckPrune(agents[x],properties,stat);
    }
}
// this is called for all agents to see if they are pruned
void GPU_Util::CheckPrune(Agent out, Properties properties, Stat stat){
    if ((out.DistanceFrom(properties.agentStartX,properties.agentStartY)/stat.d_avg + out.Energy(properties.gravity, properties.friction)/stat.E_avg) - stat.offset <=0) {
        out.pruned = true;
        // also do we need a counter for the total number of points pruned?
        //prune_counter += 1:
    ///////////////// do we need to do something here to make the list/ directory that we discussed with adi
    }
}
//Must have a non null out agent
Agent GPU_Util::AgentStep(Agent in, float newDirection, Properties properties, Map map){
    //printf("Before\n");
    Agent out;
    out.pruned = false;
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
    
    out = AgentTravel(in,out,newDirection,properties,map);
    //printf("After %f, %f, %f, %f\n",out.positionX,out.positionY,out.time,out.direction);
    out = AgentHeight(in,out,newDirection,properties,map);
    //printf("After Again\n");
    return out;
    
}

Agent GPU_Util::AgentTravel(Agent in, Agent out, float newDirection, Properties properties, Map map){  
    out.positionX = in.positionX + cos(newDirection) * properties.travelDistance;
    out.positionY = in.positionY + sin(newDirection) * properties.travelDistance;
    //printf("Agent position %f,%f\n",out->positionX,out->positionY);
    out.direction = newDirection;
    return out;
    
}
//must have out positionX and positionY populated
Agent GPU_Util::AgentHeight(Agent in, Agent out, float newDirection, Properties properties, Map map){
    
    out.height = map.GetHeight(out.positionX,out.positionY);
    if(isnan(out.height) || (2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) < 0){
        //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
        //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
        //printf("Less than zero\n");
        out.pruned = true;
        return out;
    }
    

    out.velocity = sqrt(2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) * properties.friction;
    out.time += properties.travelDistance/out.velocity;
    //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
    return out;
}
