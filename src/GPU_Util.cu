#include "hip/hip_runtime.h"
#include "Agent.h"
#include "Map.h"
#include "Properties.h"
#include "math.h"
#include "GPU_Util.h"
#include "random"


int intRandGPU(const int & min, const int & max) {
    static thread_local std::mt19937 generator;
    std::uniform_int_distribution<int> distribution(min,max);
    return distribution(generator);
}
void SwapValueGPU(Agent &a, Agent &b) {
   Agent t = a;
   a = b;
   b = t;
}

void ShuffleGPU(Agent* agents, int count){

    for(int x = 0;x<count;x++){
        int index1 = intRandGPU(0,count-1);
        int index2 = intRandGPU(0,count-1);
        SwapValueGPU(agents[index1],agents[index2]);

    }

}

void GPU_Util::StepAll(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map){
    for(int x = 0;x<inCount;x++){
		int aIndex = x*properties.numberOfDirectionSpawn;
		for(int y = 0;y<properties.numberOfDirectionSpawn;y++){
			float newDirection = in[x].direction - properties.directionSpawnRadius/2 + properties.directionSpawnRadius/(properties.numberOfDirectionSpawn-1) * y;
			//a[aIndex+y] = Agent();
			out[aIndex+y] = AgentStep(in[x],newDirection,properties,map);		
			//printf("Agent position %f,%f\n",a[aIndex+y].positionX,a[aIndex+y].positionY);
		}
	}
}
//this will be done in serial


void GPU_Util::Prune(Agent* agents,Agent* out,long count, long amountToPrune){
    srand (100);
    long x = 0;
    ShuffleGPU(agents,count);
    for(int i=0;i<count;i++){
        agents[x].pruned = true;
        //printf("ID pruned: %ld\n", x);
    }
}




//Must have a non null out agent
Agent GPU_Util::AgentStep(Agent in, float newDirection, Properties properties, Map map){
    //printf("Before\n");
    Agent out;
    out.pruned = false;
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
    
    out = AgentTravel(in,out,newDirection,properties,map);
    //printf("After %f, %f, %f, %f\n",out.positionX,out.positionY,out.time,out.direction);
    out = AgentHeight(in,out,newDirection,properties,map);
    //printf("After Again\n");
    return out;
    
}

Agent GPU_Util::AgentTravel(Agent in, Agent out, float newDirection, Properties properties, Map map){  
    out.positionX = in.positionX + cos(newDirection) * properties.travelDistance;
    out.positionY = in.positionY + sin(newDirection) * properties.travelDistance;
    //printf("Agent position %f,%f\n",out->positionX,out->positionY);
    out.direction = newDirection;
    return out;
    
}
//must have out positionX and positionY populated
Agent GPU_Util::AgentHeight(Agent in, Agent out, float newDirection, Properties properties, Map map){
    
    out.height = map.GetHeight(out.positionX,out.positionY);
    if(isnan(out.height) || (2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) < 0){
        //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
        //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
        //printf("Less than zero\n");
        out.pruned = true;
        return out;
    }
    

    out.velocity = sqrt(2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) * properties.friction;
    out.time += properties.travelDistance/out.velocity;
    //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
    return out;
}
