#include "hip/hip_runtime.h"
#include "Agent.h"
#include "Map.h"
#include "Properties.h"
#include "math.h"
#include "GPU_Util.h"
#include "random"


__global__
void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map);
int intRandGPU(const int & min, const int & max) {
    static thread_local std::mt19937 generator;
    std::uniform_int_distribution<int> distribution(min,max);
    return distribution(generator);
}
void SwapValueGPU(Agent &a, Agent &b) {
   Agent t = a;
   a = b;
   b = t;
}

void ShuffleGPU(Agent* agents, int count){

    for(int x = 0;x<count;x++){
        int index1 = intRandGPU(0,count-1);
        int index2 = intRandGPU(0,count-1);
        SwapValueGPU(agents[index1],agents[index2]);

    }

}

__global__ void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, Properties* properties, Map* map){
    int x = blockDim.x;
    int y = blockDim.y;
    int z = blockDim.z;

    int x_b = blockIdx.x;
    int y_b = blockIdx.y;
    int z_b = blockIdx.z;
    
    int x_t = threadIdx.x;
    int y_t = threadIdx.y;
    int z_t = threadIdx.z;

    //int mainId = 
    //printf("x:%i,y:%i,z:%i  x_t:%i,y_t:%i,z_y:%i\n",x,y,z,x_t,y_t,z_t);


}
void GPU_Util::StepAll(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map){

    //10
    //10
    //10
    printf("TETSSTSTS hh\n");
    dim3 DimGrid(1,1,1);
    dim3 DimBlock(32,1,1);

    Agent* in_d;
    Agent* out_d;
    Properties* properties_d;
    Map* map_d;
    hipMalloc((void **)&properties_d,sizeof(Properties));
    hipMalloc((void **)&map_d,sizeof(Map));
    hipMalloc((void **)&out_d,outCount*sizeof(Agent));
    hipMalloc((void **)&in_d,inCount*sizeof(Agent));
    hipMemcpy(map_d,&map,sizeof(Map),hipMemcpyHostToDevice);
    hipMemcpy(properties_d,&properties,sizeof(Properties),hipMemcpyHostToDevice);
    hipMemcpy(in_d,in,inCount*sizeof(Agent),hipMemcpyHostToDevice);

    GPU_Step<<<DimGrid,DimBlock>>>(in_d,inCount,out_d,outCount,properties_d,map_d);

    hipDeviceSynchronize();
    printf("Cuda done\n");
    for(int x = 0;x<inCount;x++){
		int aIndex = x*properties.numberOfDirectionSpawn;
		for(int y = 0;y<properties.numberOfDirectionSpawn;y++){
			float newDirection = in[x].direction - properties.directionSpawnRadius/2 + properties.directionSpawnRadius/(properties.numberOfDirectionSpawn-1) * y;
			//a[aIndex+y] = Agent();
			out[aIndex+y] = AgentStep(in[x],newDirection,properties,map);		
			//printf("Agent position %f,%f\n",a[aIndex+y].positionX,a[aIndex+y].positionY);
		}
	}
}



void GPU_Util::Prune(Agent* agents,Agent* out,long count, long amountToPrune){
    srand (100);
    long x = 0;
    ShuffleGPU(agents,count);
    for(int i=0;i<count;i++){
        agents[x].pruned = true;
        //printf("ID pruned: %ld\n", x);
    }
}




//Must have a non null out agent

__device__ Agent GPU_Util::AgentStep(Agent in, float newDirection, Properties properties, Map map){
    //printf("Before\n");
    Agent out;
    out.pruned = false;
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
    
    out = AgentTravel(in,out,newDirection,properties,map);
    //printf("After %f, %f, %f, %f\n",out.positionX,out.positionY,out.time,out.direction);
    out = AgentHeight(in,out,newDirection,properties,map);
    //printf("After Again\n");
    return out;
    
}

__device__ Agent GPU_Util::AgentTravel(Agent in, Agent out, float newDirection, Properties properties, Map map){  
    out.positionX = in.positionX + cos(newDirection) * properties.travelDistance;
    out.positionY = in.positionY + sin(newDirection) * properties.travelDistance;
    //printf("Agent position %f,%f\n",out->positionX,out->positionY);
    out.direction = newDirection;
    return out;
    
}
//must have out positionX and positionY populated

__device__ Agent GPU_Util::AgentHeight(Agent in, Agent out, float newDirection, Properties properties, Map map){
    
    out.height = map.GetHeight(out.positionX,out.positionY);
    if(isnan(out.height) || (2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) < 0){
        //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
        //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
        //printf("Less than zero\n");
        out.pruned = true;
        return out;
    }
    

    out.velocity = sqrt(2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) * properties.friction;
    out.time += properties.travelDistance/out.velocity;
    //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
    return out;
}
