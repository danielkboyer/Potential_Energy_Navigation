#include "hip/hip_runtime.h"
#include "Agent.h"
#include "Map.h"
#include "Properties.h"
#include "math.h"
#include "GPU_Util.h"
#include "random"
#include <fstream>
#include "Point.h"
#include "Agent.h"
#include "math.h"
#include "stdio.h"
using namespace std;



__host__ __device__ Agent::Agent(){}
__host__ __device__ Agent::Agent(Agent& agent){
    //printf("Constructor called\n");
    direction = agent.direction;
    positionX = agent.positionX;
    positionY = agent.positionY;
    height = agent.height;
    velocity = agent.velocity;
    time = agent.time;
    Id = agent.Id;
    parentId = agent.parentId;
    pruned = agent.pruned;
    percentage = agent.percentage;
}
__host__ __device__ Agent::Agent(float direction, float positionX, float positionY, float height, float velocity, float time, int Id, int parentId,float percentage,bool pruned = false):direction(direction),positionX(positionX),positionY(positionY),
height(height),velocity(velocity),time(time),Id(Id),parentId(parentId),pruned(pruned),percentage(percentage){
}


__host__ __device__ float Agent::DistanceFrom(float x, float y){
    return sqrt((x-positionX)*(x-positionX) + (y-positionY)*(y-positionY));
}
        
__host__ __device__ float Agent::Energy(float gravity, float friction){
    return ((velocity)*(velocity)/2 + (height)*(gravity))*friction;
}    

__global__
void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map);

//__device__
int intRandGPU(const int & min, const int & max) {
    static thread_local std::mt19937 generator;
    std::uniform_int_distribution<int> distribution(min,max);
    return distribution(generator);
}
//__device__
void SwapValueGPU(Agent &a, Agent &b) {
   Agent t = a;
   a = b;
   b = t;
}
//__device__
void ShuffleGPU(Agent* agents, int count){

    for(int x = 0;x<count;x++){
        int index1 = intRandGPU(0,count-1);
        int index2 = intRandGPU(0,count-1);
        SwapValueGPU(agents[index1],agents[index2]);

    }

}

__global__ void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, int apt, Properties* properties, Map* map){

    int x = blockDim.x;

    int x_b = blockIdx.x;
    int y_b = blockIdx.y;
   
    
    int x_t = threadIdx.x;

    int mainId = (x_b * 512)+(y_b*x*512)+x_t;
    int outId =mainId * properties->numberOfDirectionSpawn;
    for(int x = 0;x<apt && (mainId*apt + x) < inCount;x++){
        printf("Looping, mainid:%i\n",mainId);
		int aIndex = x*properties->numberOfDirectionSpawn;
        //printf("Number of direction spawn %f\n",properties->numberOfDirectionSpawn);
        //printf("Travel Distance %f\n",properties->travelDistance);
		for(int y = 0;y<(int)properties->numberOfDirectionSpawn;y++){
            
			float newDirection = in[x].direction - properties->directionSpawnRadius/2 + properties->directionSpawnRadius/(properties->numberOfDirectionSpawn-1) * y;
			
            int inIndex = mainId + x;
            int outIndex = outId + aIndex + y;
            
            printf("Creating agent. OutIndex: %i\n",outIndex);
            Agent outAgent;
            outAgent.pruned = false;
            Agent inAgent = in[inIndex];
            //printf("In Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",inAgent.positionX,inAgent.positionY,inAgent.velocity,inAgent.height,properties->gravity,properties->friction);
    
            outAgent.positionX = inAgent.positionX + cos(newDirection) * properties->travelDistance;
            outAgent.positionY = inAgent.positionY + sin(newDirection) * properties->travelDistance;
            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
    
            outAgent.direction = newDirection;

            //Get the height
            int startX = floor(inAgent.positionX/map->_pointDistance);
            int startY = floor(inAgent.positionY/map->_pointDistance);
            //printf("StartX: %d , StartY: %d\n",startX,startY);
            if(startX < 0)
                outAgent.height = NAN;
            if(startX +1 >= map->_width)
                outAgent.height = NAN;
            if(startY -1 < 0)
                outAgent.height = NAN;
            if(startY >= map->_height)
                outAgent.height = NAN;
            //printf("x %f, y %f\n",x/_pointDistance,y/_pointDistance);
            float xPoint = x/map->_pointDistance - startX;
            float yPoint = y/map->_pointDistance - startY;
            //printf("yPoint %f, xPoint %f\n",yPoint,xPoint);
            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
            //printf("Map Pointe distance %f\n",map->_pointDistance);
            //printf("Map Pointe %f\n",points[0].height);
            for(int t = 0;t<map->_width;t++){
                printf("GPU Map: %f\n",map->points[t].height);
            }
            // outAgent.height = (map->_pointDistance - xPoint)*(map->_pointDistance - yPoint)*(points[startY][startX].height) + 
            //         (map->_pointDistance)*(map->_pointDistance - yPoint)*(points[startY][startX+1].height) +
            //         (map->_pointDistance - xPoint)*(map->_pointDistance)*(points[startY-1][startX].height) +
            //         (map->_pointDistance)*(map->_pointDistance)*(points[startY-1][startX+1].height);

            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);

            //done getting height
            if(isnan(outAgent.height) || (2*properties->gravity*(inAgent.height - outAgent.height) + inAgent.velocity*inAgent.velocity) < 0){
              
                outAgent.pruned = true;
                //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
    
                out[outIndex] = outAgent;
                continue;
            }
    

            outAgent.velocity = sqrt(2*properties->gravity*(inAgent.height - outAgent.height) + inAgent.velocity*inAgent.velocity) * properties->friction;
            outAgent.time += properties->travelDistance/outAgent.velocity;
            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
    
            out[outIndex] = outAgent;

        }
        
    }

    //printf("x:%i,y:%i,z:%i  x_t:%i,y_t:%i,z_y:%i\n",x,y,z,x_t,y_t,z_t);


}
void GPU_Util::StepAll(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map){

    //10
    //10
    //10
    int atp = 50;
    int gridNumber = (int)ceil(sqrt(((float)inCount/(float)atp))/(float)512);
    dim3 DimGrid(gridNumber,gridNumber,1);
    dim3 DimBlock(512,1,1);

    Agent* in_d;
    Agent* out_d;
    Properties* properties_d;
    Map* map_d;


    printf("PROPERTIES : NUMBER %f\n",properties.numberOfDirectionSpawn);


    Point* pointer;

    hipMalloc((void **)&pointer->points,sizeof(Point *));
    ///POINTS MEMORY
    hipMalloc((void **)&map_d,sizeof(Map));
    hipMemcpy(map_d,&map,sizeof(Map),hipMemcpyHostToDevice);

    Point *temp_data;
    hipMalloc((void **)&(temp_data),sizeof(Point)*map._width*map._height);
    hipMemcpy(&(map_d->points),&(temp_data),sizeof(Point)*map._width*map._height,hipMemcpyHostToDevice);

    hipMemcpy(temp_data,map.points,sizeof(Point)*map._width*map._height,hipMemcpyHostToDevice);
    

    //End Point Memory
    hipMalloc((void **)&properties_d,sizeof(Properties));

    hipMalloc((void **)&out_d,outCount*sizeof(Agent));
    hipMalloc((void **)&in_d,inCount*sizeof(Agent));
    for(int x = 0;x<map._width*map._height;x++){
        //printf("Map at %i,%f\n",x,points_h[x].height);
    }
    printf("Values %i,%i,%i\n",map._width,map._height,map._width*map._height);
    
    
    printf("Got Here\n");
    hipMemcpy(properties_d,&properties,sizeof(Properties),hipMemcpyHostToDevice);
    printf("Got Here\n");
    hipMemcpy(in_d,in,inCount*sizeof(Agent),hipMemcpyHostToDevice);
    printf("Got Here\n");
    GPU_Step<<<DimGrid,DimBlock>>>(in_d,inCount,out_d,outCount,atp,properties_d,map_d);
    hipDeviceSynchronize();
    hipMemcpy(out,out_d,outCount*sizeof(Agent),hipMemcpyDeviceToHost);
    
    for(int x = 0;x<outCount;x++){
        printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out[x].positionX,out[x].positionY,out[x].velocity,out[x].height,properties.gravity,properties.friction);
    
    }
}



void GPU_Util::Prune(Agent* agents,Agent* out,long count, long amountToPrune){
    long keepAmount = count - amountToPrune;
    ShuffleGPU(agents,count);
    vector<int> good;
    vector<int> bad;
    for(int x = 0;x<count;x++){
        if(isnan(agents[x].velocity) || agents[x].velocity <= 0 || agents[x].pruned == true){
            agents[x].pruned = true;
            bad.push_back(x);
        }
        else{
            good.push_back(x);
        }
    }
    for(int x =0 ;x<keepAmount;x++){
        if(x >= good.size()){
            out[x] = agents[bad[x-good.size()]];
            continue;
        }
        out[x] = agents[good[x]];
    }
    out[0].percentage = ((float)(bad.size()))/(float)count;
}




//Must have a non null out agent

Agent GPU_Util::AgentStep(Agent in, float newDirection, Properties properties, Map map){
    //printf("Before\n");
    Agent out;
    out.pruned = false;
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
    
    out = AgentTravel(in,out,newDirection,properties,map);
    //printf("After %f, %f, %f, %f\n",out.positionX,out.positionY,out.time,out.direction);
    out = AgentHeight(in,out,newDirection,properties,map);
    //printf("After Again\n");
    return out;
    
}

Agent GPU_Util::AgentTravel(Agent in, Agent out, float newDirection, Properties properties, Map map){  
    out.positionX = in.positionX + cos(newDirection) * properties.travelDistance;
    out.positionY = in.positionY + sin(newDirection) * properties.travelDistance;
    //printf("Agent position %f,%f\n",out->positionX,out->positionY);
    out.direction = newDirection;
    return out;
    
}
//must have out positionX and positionY populated

Agent GPU_Util::AgentHeight(Agent in, Agent out, float newDirection, Properties properties, Map map){
    
    out.height = map.GetHeight(out.positionX,out.positionY);
    if(isnan(out.height) || (2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) < 0){
        //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
        //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
        //printf("Less than zero\n");
        out.pruned = true;
        return out;
    }
    

    out.velocity = sqrt(2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) * properties.friction;
    out.time += properties.travelDistance/out.velocity;
    //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
    return out;
}
