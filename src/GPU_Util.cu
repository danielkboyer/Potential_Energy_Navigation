#include "hip/hip_runtime.h"
#include "Agent.h"
#include "Map.h"
#include "Properties.h"
#include "math.h"
#include "GPU_Util.h"
#include "random"
#include <fstream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h> 
#include "Point.h"
#include "Agent.h"
#include "math.h"
#include "stdio.h"
#include "thrust/device_vector.h"

using namespace std;



__host__ __device__ Agent::Agent(){}
__host__ __device__ Agent::Agent(Agent& agent){
    //printf("Constructor called\n");
    direction = agent.direction;
    positionX = agent.positionX;
    positionY = agent.positionY;
    height = agent.height;
    velocity = agent.velocity;
    time = agent.time;
    Id = agent.Id;
    parentId = agent.parentId;
    pruned = agent.pruned;
    percentage = agent.percentage;
}
__host__ __device__ Agent::Agent(float direction, float positionX, float positionY, float height, float velocity, float time, int Id, int parentId,float percentage,bool pruned = false):direction(direction),positionX(positionX),positionY(positionY),
height(height),velocity(velocity),time(time),Id(Id),parentId(parentId),pruned(pruned),percentage(percentage){
}


__host__ __device__ float Agent::DistanceFrom(float x, float y){
    return sqrt((x-positionX)*(x-positionX) + (y-positionY)*(y-positionY));
}
        
__host__ __device__ float Agent::Energy(float gravity, float friction){
    return ((velocity)*(velocity)/2 + (height)*(gravity))*friction;
}    

__global__
void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map);

__device__
int intRandGPU(int id,const int & min, const int & max) {
    hiprandState state;
    hiprand_init((unsigned long long)clock() + id, 0, 0, &state);
    double rand1 = (hiprand_uniform_double(&state)) *(max-min+0.999999);
    return (int)truncf(rand1);
}
__device__
void SwapValueGPU(Agent &a, Agent &b) {
   Agent t = a;
   a = b;
   b = t;
}
__device__
void ShuffleGPU(Agent* agents, int count, int id){

    for(int x = 0;x<count;x++){
        int index1 = intRandGPU(id,0,count-1);
        int index2 = intRandGPU(id,0,count-1);
        SwapValueGPU(agents[index1],agents[index2]);

    }

}

__global__ void GPU_Step(Agent* in, int inCount, Agent* out, int outCount, int apt, Properties* properties, Map* map,Point* points){

    int x = blockDim.x;

    int x_b = blockIdx.x;
    int y_b = blockIdx.y;
   
    
    int x_t = threadIdx.x;

    int mainId = ((x_b * 512)+(y_b*x*512)+x_t)*apt;
    //printf("Main ID: %i\n",mainId);
    int outId =mainId * properties->numberOfDirectionSpawn;
    for(int x = 0;x<apt && (outId+ x) < outCount;x++){
        //printf("Looping, mainid:%i\n",mainId);
		int aIndex = x*properties->numberOfDirectionSpawn;
        //printf("Number of direction spawn %f\n",properties->numberOfDirectionSpawn);
        //printf("Travel Distance %f\n",properties->travelDistance);
		for(int y = 0;y<(int)properties->numberOfDirectionSpawn;y++){
            
            int inIndex = mainId + x;
            int outIndex = outId + aIndex + y;
            
            //printf("Main ID: %i, OutIndex: %i, InIndex: %i\n",mainId,outIndex,inIndex);
            Agent outAgent;
            outAgent.pruned = false;
            Agent inAgent = in[inIndex];
            if(inAgent.pruned == true){
                outAgent.pruned = true;
                out[outIndex] = outAgent;
                continue;
            }
			float newDirection = inAgent.direction - properties->directionSpawnRadius/2 + properties->directionSpawnRadius/(properties->numberOfDirectionSpawn-1) * y;
			
            
            //printf("In Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",inAgent.positionX,inAgent.positionY,inAgent.velocity,inAgent.height,properties->gravity,properties->friction);
    
            outAgent.positionX = inAgent.positionX + cos(newDirection) * properties->travelDistance;
            outAgent.positionY = inAgent.positionY + sin(newDirection) * properties->travelDistance;
            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
    
            outAgent.direction = newDirection;

            //Get the height
            int startX = floor(inAgent.positionX/map->_pointDistance);
            int startY = floor(inAgent.positionY/map->_pointDistance);
            //printf("StartX: %d , StartY: %d\n",startX,startY);

            if(startX < 0 || startX +1 >= map->_width || startY -1 < 0 || startY >= map->_height){
                outAgent.height = NAN;
                outAgent.pruned = true;
                
                out[outIndex] = outAgent;
                continue;
            }
            float xPoint = inAgent.positionX/map->_pointDistance - startX;
            float yPoint = inAgent.positionY/map->_pointDistance - startY;

            outAgent.height = (map->_pointDistance - xPoint)*(map->_pointDistance - yPoint)*(points[startY*map->_width+startX].height) + 
                    (map->_pointDistance)*(map->_pointDistance - yPoint)*(points[startY*map->_width+ startX+1].height) +
                    (map->_pointDistance - xPoint)*(map->_pointDistance)*(points[(startY-1)*map->_width+ startX].height) +
                    (map->_pointDistance)*(map->_pointDistance)*(points[(startY-1)*map->_width + startX+1].height);

            

            //done getting height
            if(isnan(outAgent.height) || (2*properties->gravity*(inAgent.height - outAgent.height) + inAgent.velocity*inAgent.velocity) < 0){
              
                outAgent.pruned = true;
                
                out[outIndex] = outAgent;
                continue;
            }
    

            outAgent.velocity = sqrt(2*properties->gravity*(inAgent.height - outAgent.height) + inAgent.velocity*inAgent.velocity) * properties->friction;
            outAgent.time += properties->travelDistance/outAgent.velocity;
            //printf("Out Agent = PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",outAgent.positionX,outAgent.positionY,outAgent.velocity,outAgent.height,properties->gravity,properties->friction);
    
            out[outIndex] = outAgent;

        }
        
    }

    //printf("x:%i,y:%i,z:%i  x_t:%i,y_t:%i,z_y:%i\n",x,y,z,x_t,y_t,z_t);


}
void GPU_Util::StepAll(Agent* in, int inCount, Agent* out, int outCount, Properties properties, Map map){

    //10
    //printf("OUT COUNT: %i\n",outCount);
    //10
    //10
    int atp = 9;
    int gridNumber = (int)ceil(((float)inCount/(float)atp)/(float)512);
    //printf("Grid Number: %i\n",gridNumber);
    dim3 DimGrid(gridNumber,gridNumber,1);
    dim3 DimBlock(512,1,1);

    Agent* in_d;
    Agent* out_d;
    Properties* properties_d;
    Map* map_d;
    Point* points_d;


    //printf("PROPERTIES : NUMBER %f\n",properties.numberOfDirectionSpawn);

    ///POINTS MEMORY
    //hipMalloc((void **)&map_d,sizeof(Map));
    //hipMemcpy(map_d,&map,sizeof(Map),hipMemcpyHostToDevice);

    hipMalloc((void **)&points_d,sizeof(Point)*map._width*map._height);
    hipMemcpy(points_d,map.points,sizeof(Point)*map._width*map._height,hipMemcpyHostToDevice);

    hipMalloc((void **)&map_d, sizeof(Map));
    hipMemcpy(map_d,&map, sizeof(Map),hipMemcpyHostToDevice);
    //hipMemcpy(temp_data,map.points,sizeof(Point)*map._width*map._height,hipMemcpyHostToDevice);
    

    //End Point Memory
    hipMalloc((void **)&properties_d,sizeof(Properties));

    hipMalloc((void **)&out_d,outCount*sizeof(Agent));
    hipMalloc((void **)&in_d,inCount*sizeof(Agent));

    
    hipMemcpy(properties_d,&properties,sizeof(Properties),hipMemcpyHostToDevice);

    hipMemcpy(in_d,in,inCount*sizeof(Agent),hipMemcpyHostToDevice);

    GPU_Step<<<DimGrid,DimBlock>>>(in_d,inCount,out_d,outCount,atp,properties_d,map_d,points_d);
    hipDeviceSynchronize();
    hipMemcpy(out,out_d,outCount*sizeof(Agent),hipMemcpyDeviceToHost);
    hipFree(in_d);
    hipFree(out_d);
    hipFree(properties_d);
    hipFree(map_d);
    hipFree(points_d);
    // for(int x = 0;x<outCount;x++){
    //     printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out[x].positionX,out[x].positionY,out[x].velocity,out[x].height,properties.gravity,properties.friction);
    // }
}


__global__ void PruneGPU(Agent* agents,Agent* out,long count,long pruneAmountTotal,long apt, long aptP){
    int x = blockDim.x;

    int x_b = blockIdx.x;
    int y_b = blockIdx.y;
   
    
    int x_t = threadIdx.x;
    long keepAmount = apt - aptP;
    int mainId = ((x_b * 512)+(y_b*x*512)+x_t)*apt;
    if(mainId < count){
        return;
    }
    int outId = ((x_b * 512)+(y_b*x*512)+x_t)*keepAmount;
    
    Agent* my_local_agents = new Agent[apt];
    for(int x = 0;x<apt;x++){
        if(mainId+x < count){
        my_local_agents[x] = agents[mainId+x];
        }
        else{
            Agent dummy;
            dummy.pruned = true;
            my_local_agents[x] = dummy;
        }
    }

    ShuffleGPU(my_local_agents,apt,mainId);
    int* good = new int[apt];
    int* bad = new int[apt];
    int goodCount = 0;
    int badCount = 0;
    for(int x = 0;x<apt;x++){
        if(isnan(my_local_agents[x].velocity) || my_local_agents[x].velocity <= 0 || my_local_agents[x].pruned == true){
            my_local_agents[x].pruned = true;
            bad[badCount++] = x;
        }
        else{
            bad[goodCount++] = x;
        }
    }
    
    for(int x =0 ;x<keepAmount;x++){
        if(x >= goodCount){
            if(mainId+x < pruneAmountTotal){
                out[outId+x] = my_local_agents[bad[x-goodCount]];
            }
            continue;
        }
        if(mainId+x < pruneAmountTotal){
            out[outId+x] = my_local_agents[good[x]];
        }
    }
    out[outId].percentage = ((float)(badCount))/(float)apt;
    delete[] my_local_agents;
    delete[] good;
    delete[] bad;
}



void GPU_Util::Prune(Agent* agents,Agent* out,long count, long amountToPrune,Properties properties){

    int apt = 16;
    int amountToAdd = count%512;
    
    int gridNumber = (int)ceil(((float)(count+amountToAdd)/(float)apt)/(float)512);
    int randomPerThread = (count+amountToAdd)/512/pow(gridNumber,2);
    //printf("Grid Number: %i\n",gridNumber);
    dim3 DimGrid(gridNumber,gridNumber,1);
    dim3 DimBlock(512,1,1);

    Agent* in_d;
    Agent* out_d;


    hipMalloc((void **)&out_d,(count-amountToPrune)*sizeof(Agent));
    hipMalloc((void **)&in_d,count*sizeof(Agent));

    
    hipMemcpy(out_d,&out,sizeof(Agent)*(count-amountToPrune),hipMemcpyHostToDevice);

    hipMemcpy(in_d,agents,count*sizeof(Agent),hipMemcpyHostToDevice);

    PruneGPU<<<DimGrid,DimBlock>>>(in_d,out_d,count,(count-amountToPrune),apt,randomPerThread);
    hipDeviceSynchronize();
    hipMemcpy(out,out_d,(count-amountToPrune)*sizeof(Agent),hipMemcpyDeviceToHost);
    hipFree(in_d);
    hipFree(out_d);
}




//Must have a non null out agent

Agent GPU_Util::AgentStep(Agent in, float newDirection, Properties properties, Map map){
    //printf("Before\n");
    Agent out;
    out.pruned = false;
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
    
    out = AgentTravel(in,out,newDirection,properties,map);
    //printf("After %f, %f, %f, %f\n",out.positionX,out.positionY,out.time,out.direction);
    out = AgentHeight(in,out,newDirection,properties,map);
    //printf("After Again\n");
    return out;
    
}

Agent GPU_Util::AgentTravel(Agent in, Agent out, float newDirection, Properties properties, Map map){  
    out.positionX = in.positionX + cos(newDirection) * properties.travelDistance;
    out.positionY = in.positionY + sin(newDirection) * properties.travelDistance;
    //printf("Agent position %f,%f\n",out->positionX,out->positionY);
    out.direction = newDirection;
    return out;
    
}
//must have out positionX and positionY populated

Agent GPU_Util::AgentHeight(Agent in, Agent out, float newDirection, Properties properties, Map map){
    
    out.height = map.GetHeight(out.positionX,out.positionY);
    if(isnan(out.height) || (2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) < 0){
        //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
        //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
        //printf("Less than zero\n");
        out.pruned = true;
        return out;
    }
    

    out.velocity = sqrt(2*properties.gravity*(in.height - out.height) + in.velocity*in.velocity) * properties.friction;
    out.time += properties.travelDistance/out.velocity;
    //printf("InHeight %f, InVelocity: %f\n",in.height,in.velocity);
    //printf("PositionX, %f, PositionY %f, Velocity %f, height %f, gravity %f, friciton %f\n",out.positionX,out.positionY,out.velocity,out.height,properties.gravity,properties.friction);
     
    return out;
}
